#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
 *
 * time:    winter term 2012/13 / March 11-18, 2013
 *
 * project: convolution
 * file:    convolution_gpu.cu
 *
 * 
 \******* PLEASE ENTER YOUR CORRECT STUDENT LOGIN, NAME AND ID BELOW *********/
const char* gpu_studentLogin = "p107";
const char* gpu_studentName = "Marco Seravalli";
const int gpu_studentID = 3626387;
/****************************************************************************\
*
 * In this file the following methods have to be edited or completed:
 *
 * gpu_convolutionGrayImage_gm_d
 * gpu_convolutionGrayImage_gm_cm_d
 * gpu_convolutionGrayImage_sm_d
 * gpu_convolutionGrayImage_sm_cm_d
 * gpu_convolutionGrayImage_dsm_cm_d
 * gpu_convolutionInterleavedRGB_dsm_cm_d
 * gpu_convolutionInterleavedRGB_tex_cm_d
 *
 \****************************************************************************/

#include "convolution_gpu.cuh"

#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <iostream>

#define TEXTURE_OFFSET      0.5f  // offset for indexing textures
#define BW                  16    // block width
#define BH                  16    // block height
#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE       ( 1+2*MAXKERNELRADIUS)*( 1+2*MAXKERNELRADIUS)
#define MAXSHAREDMEMSIZE    (BW+2*MAXKERNELRADIUS)*(BH+2*MAXKERNELRADIUS)

#if (MAXSHAREDMEMSIZE > 4000)   // Note: MAXSHAREDMEMSIZE <= 4000 should hold for most graphic cards to work
#error "This program will most likely not run properly because of insufficient shared memory, please reduce BW/BH/MAXKERNELRADIUS!"
#endif

// constant memory block on device
__constant__ float constKernel[MAXKERNELSIZE];

// texture memory and descriptor
hipChannelFormatDesc tex_Image_desc = hipCreateChannelDesc<float> ();
texture<float, 2, hipReadModeElementType> tex_Image;

hipChannelFormatDesc tex_Image_descF4 = hipCreateChannelDesc<float4> ();
texture<float4, 2, hipReadModeElementType> tex_ImageF4;

const char* gpu_getStudentLogin() {
  return gpu_studentLogin;
}
;
const char* gpu_getStudentName() {
  return gpu_studentName;
}
;
int gpu_getStudentID() {
  return gpu_studentID;
}
;
bool gpu_checkStudentData() {
  return strcmp(gpu_studentLogin, "p010") != 0 && strcmp(gpu_studentName,
      "John Doe") != 0 && gpu_studentID != 1234567;
}
;
bool gpu_checkStudentNameAndID() {
  return strcmp(gpu_studentName, "John Doe") != 0 && gpu_studentID != 1234567;
}
;

//----------------------------------------------------------------------------
// Gray Image Functions
//----------------------------------------------------------------------------


// mode 1 (gray): using global memory only
__global__ void gpu_convolutionGrayImage_gm_d(const float *inputImage,
    const float *kernel, float *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, size_t iPitch, size_t kPitch) {

  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;

  int i_kern, j_kern;
  int x, y;
  float tmp = 0;
  
  int i_img = blockIdx.x*blockDim.x + threadIdx.x;
  int j_img = blockIdx.y*blockDim.y + threadIdx.y;

  // ### implement a convolution ###
  if (i_img >= 0 && i_img < iWidth && j_img >= 0 && j_img < iHeight){
    for (i_kern = 0; i_kern < kWidth; ++i_kern) {
      for (j_kern = 0; j_kern < kHeight; ++j_kern) {
        x = i_img + (i_kern - (kWidth / 2));
        y = j_img + (j_kern - (kHeight / 2));
        
        if (x < 0) {
          x = 0;
        }
        else if (x >= iWidth) {
          x = iWidth - 1;
        }
        if (y < 0) {
      	  y = 0;
        }
        else if (y >= iHeight) {
          y = iHeight - 1;
        }
        tmp += kernel[j_kern * kPitch + i_kern] * inputImage[y * iPitch + x];
      }
    }
    outputImage[j_img * iPitch + i_img] = tmp;
  }

}

// mode 2 (gray): using global memory and constant memory
__global__ void gpu_convolutionGrayImage_gm_cm_d(const float *inputImage,
  float *outputImage, int iWidth, int iHeight, int kRadiusX,
  int kRadiusY, size_t iPitch) {
  
  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;
  
  int i_kern, j_kern;
  int x, y;
  float tmp = 0;
  
  int i_img = blockIdx.x*blockDim.x + threadIdx.x;
  int j_img = blockIdx.y*blockDim.y + threadIdx.y;
  
  // ### implement a convolution ###
  if (i_img >= 0 && i_img < iWidth && j_img >= 0 && j_img < iHeight){
    for (i_kern = 0; i_kern < kWidth; ++i_kern) {
      for (j_kern = 0; j_kern < kHeight; ++j_kern) {
        x = i_img + (i_kern - (kWidth / 2));
        y = j_img + (j_kern - (kHeight / 2));
        
        if (x < 0) {
          x = 0;
        }
        else if (x >= iWidth) {
          x = iWidth - 1;
        }
        if (y < 0) {
      	  y = 0;
        }
        else if (y >= iHeight) {
          y = iHeight - 1;
        }
        tmp += constKernel[j_kern * kWidth + i_kern] * inputImage[y * iPitch + x];
      }
    }
    outputImage[j_img * iPitch + i_img] = tmp;
  }

}

// mode 3 (gray): using shared memory for image and global memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_d(const float *inputImage,
    const float *kernel, float *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, size_t iPitch, size_t kPitch) {
  // make use of the constant MAXSHAREDMEMSIZE in order to define the shared memory size

  __shared__ float tile[MAXSHAREDMEMSIZE];
  const int k_w = (kRadiusX << 1)  + 1;
  const int k_h = (kRadiusY << 1) + 1;
  
  const int t_w = blockDim.x + (2 * kRadiusX);
  const int t_h = blockDim.y + (2 * kRadiusY);
  const int t_tot = t_w * t_h;

  const int global_x = blockIdx.x*blockDim.x + threadIdx.x;
  const int global_y = blockIdx.y*blockDim.y + threadIdx.y;
  const int global_idx = global_y * iPitch + global_x;
  
  const int threads = blockDim.x * blockDim.y;
  
  const int loops = (int)ceilf((float)t_tot / (float)threads);
  
  const int local_idx = threadIdx.y * blockDim.x + threadIdx.x;

  const int block_top =  blockIdx.x*blockDim.x;
  const int block_left = blockIdx.y*blockDim.y;

  int t_x, t_y, i_x, i_y;
  
  int t_idx = 0;
  
  for (int i = 0; i < loops; ++i) {
    t_idx = (local_idx + i * threads);
    if (t_idx >= 0 && t_idx < t_tot) {
      t_x = t_idx % t_w - kRadiusX;
      t_y = t_idx / t_h - kRadiusY;
      
      i_x = block_top  + t_x;
      i_y = block_left + t_y;
      
      if (i_x < 0) {
        i_x = 0;
      } else if (i_x >= iWidth) {
        i_x = iWidth - 1;
      }
      if (i_y < 0) {
        i_y = 0;
      } else if (i_y >= iHeight) {
        i_y = iHeight - 1;
      }
      
      tile[t_idx] = inputImage[i_y*iPitch + i_x];
    }
  }
  
  __syncthreads();
  
//  outputImage[global_idx] = 0;
//  outputImage[global_idx] = tile[(kRadiusY + threadIdx.y)*t_w + kRadiusX + threadIdx.x];

  int x, y;
  float tmp = 0;
  if (global_x >= 0 && global_x < iWidth && global_y >= 0 && global_y < iHeight){
    for (int i_kern = 0; i_kern < k_w; ++i_kern) {
      for (int j_kern = 0; j_kern < k_h; ++j_kern) {
        x = kRadiusX + threadIdx.x + (i_kern - (k_w / 2));
        y = kRadiusY + threadIdx.y + (j_kern - (k_h / 2));
        
        tmp += kernel[j_kern * kPitch + i_kern] * tile[y * t_w + x];
      }
    }
    outputImage[global_idx] = tmp;
  }
  
}

// mode 4 (gray): using shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_cm_d(const float *inputImage,
    float *outputImage, int iWidth, int iHeight, int kRadiusX,
    int kRadiusY, size_t iPitch) {
  // make use of the constant MAXSHAREDMEMSIZE in order to define the shared memory size

  // ### implement me ### 

}

// mode 5 (gray): using dynamically allocated shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_dsm_cm_d(const float *inputImage,
    float *outputImage, int iWidth, int iHeight, int kRadiusX,
    int kRadiusY, size_t iPitch) {

  // ### implement me ###  

}

// mode 6 (gray): using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_tex_cm_d(float *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY, size_t iPitch) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= iWidth || y >= iHeight)
    return;

  const float xx = (float) (x - kRadiusX) + TEXTURE_OFFSET;
  const float yy = (float) (y - kRadiusY) + TEXTURE_OFFSET;
  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;
  float value = 0.0f;

  for (int yk = 0; yk < kHeight; yk++)
    for (int xk = 0; xk < kWidth; xk++)
      value += tex2D(tex_Image, xx - xk, yy - yk) * constKernel[yk
          * kWidth + xk];

  outputImage[y * iPitch + x] = value;
}

void gpu_convolutionGrayImage(const float *inputImage, const float *kernel,
    float *outputImage, int iWidth, int iHeight, int kRadiusX,
    int kRadiusY, int mode) {
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  float *d_inputImage;
  float *d_kernel;
  float *d_outputImage;

  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;

  assert(kWidth * kHeight <= MAXKERNELSIZE);

  // allocate device memory
  cutilSafeCall(
      hipMallocPitch((void**) &d_inputImage, &iPitchBytes,
          iWidth * sizeof(float), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &d_outputImage, &iPitchBytes,
          iWidth * sizeof(float), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &d_kernel, &kPitchBytes,
          kWidth * sizeof(float), kHeight));
  iPitch = iPitchBytes / sizeof(float);
  kPitch = kPitchBytes / sizeof(float);
  //std::cout << "iPitchBytes=" << iPitchBytes << " iPitch=" << iPitch << " kPitchBytes=" << kPitchBytes << " kPitch=" << kPitch << std::endl;

  cutilSafeCall(
      hipMemcpy2D(d_inputImage, iPitchBytes, inputImage,
          iWidth * sizeof(float), iWidth * sizeof(float), iHeight,
          hipMemcpyHostToDevice));
  cutilSafeCall(
      hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth * sizeof(float),
          kWidth * sizeof(float), kHeight, hipMemcpyHostToDevice));

  gpu_bindConstantMemory(kernel, kWidth * kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  dim3 blockSize(BW, BH);
  dim3 gridSize(((iWidth % BW) ? (iWidth / BW + 1) : (iWidth / BW)),
      ((iHeight % BH) ? (iHeight / BH + 1) : (iHeight / BH)));

  // invoke the kernel of your choice here
  const int smSize = (blockSize.x + (kRadiusX << 1)) * (blockSize.y
      + (kRadiusY << 1)) * sizeof(float);

  switch (mode) {
  case 1:
    gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    break;
  case 2:
    gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    break;
  case 3:
    gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    break;
  case 4:
    gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    break;
  case 5:
    gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    break;
  case 6:
    gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize>>>(d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    break;
  default:
    std::cout << "gpu_convolutionGrayImage() Warning: mode " << mode
        << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize());
  cutilSafeCall(
      hipMemcpy2D(outputImage, iWidth * sizeof(float), d_outputImage,
          iPitchBytes, iWidth * sizeof(float), iHeight,
          hipMemcpyDeviceToHost));

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall(hipFree(d_inputImage));
  cutilSafeCall(hipFree(d_outputImage));
  cutilSafeCall(hipFree(d_kernel));
}

//----------------------------------------------------------------------------
// RGB Image Functions (for separated color channels)
//----------------------------------------------------------------------------


void gpu_convolutionRGB(const float *inputImage, const float *kernel,
    float *outputImage, int iWidth, int iHeight, int kRadiusX,
    int kRadiusY, int mode) {
  const int imgSize = iWidth * iHeight;
  gpu_convolutionGrayImage(inputImage, kernel, outputImage, iWidth, iHeight,
      kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage + imgSize, kernel,
      outputImage + imgSize, iWidth, iHeight, kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage + (imgSize << 1), kernel,
      outputImage + (imgSize << 1), iWidth, iHeight, kRadiusX, kRadiusY,
      mode);
}

//----------------------------------------------------------------------------
// RGB Image Functions (for interleaved color channels)
//----------------------------------------------------------------------------


// mode 5 (interleaved): using dynamically allocated shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_dsm_cm_d(
    const float3 *inputImage, float3 *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, size_t iPitchBytes) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  float3 value = make_float3(0.0f, 0.0f, 0.0f);

  // ### implement me ### 

  *((float3*) (((char*) outputImage) + y * iPitchBytes) + x) = value;
}

__global__ void gpu_ImageFloat3ToFloat4_d(const float3 *inputImage,
    float4 *outputImage, int iWidth, int iHeight, size_t iPitchBytes,
    size_t oPitchBytes) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= iWidth || y >= iHeight)
    return;

  float3 rgb = *((float3*) ((char*) inputImage + y * iPitchBytes) + x);
  *((float4*) (((char*) outputImage) + y * oPitchBytes) + x) = make_float4(
      rgb.x, rgb.y, rgb.z, 0.0f);
}

// mode 6 (interleaved): using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_tex_cm_d(float3 *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY, size_t oPitchBytes) {

  // ### implement me ### 

}

void gpu_convolutionInterleavedRGB(const float *inputImage,
    const float *kernel, float *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, int mode) {
  size_t iPitchBytesF3, iPitchBytesF4;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;

  //  allocate memory and copy data
  cutilSafeCall(
      hipMallocPitch((void**) &(d_inputImageF3), &iPitchBytesF3,
          iWidth * sizeof(float3), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_outputImageF3), &iPitchBytesF3,
          iWidth * sizeof(float3), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_inputImageF4), &iPitchBytesF4,
          iWidth * sizeof(float4), iHeight));

  cutilSafeCall(
      hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage,
          iWidth * sizeof(float3), iWidth * sizeof(float3), iHeight,
          hipMemcpyHostToDevice));

  dim3 blockSize(BW, BH);
  dim3 gridSize(((iWidth % BW) ? (iWidth / BW + 1) : (iWidth / BW)),
      ((iHeight % BH) ? (iHeight / BH + 1) : (iHeight / BH)));
  int smSizeF3 = (blockSize.x + (kRadiusX << 1)) * (blockSize.y + (kRadiusY
      << 1)) * sizeof(float3);

  // convert image from float3* to float4*
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);

  gpu_bindConstantMemory(kernel, kWidth * kHeight);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);

  switch (mode) {
  case 1:
  case 2:
  case 3:
  case 4:
    std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode
        << " is not supported." << std::endl;
    break;
  case 5:
    gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    break;
  case 6:
    gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    break;
  default:
    std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode
        << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize());
  cutilSafeCall(
      hipMemcpy2D(outputImage, iWidth * sizeof(float3), d_outputImageF3,
          iPitchBytesF3, iWidth * sizeof(float3), iHeight,
          hipMemcpyDeviceToHost));

  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall(hipFree(d_inputImageF4));
  cutilSafeCall(hipFree(d_inputImageF3));
  cutilSafeCall(hipFree(d_outputImageF3));
}

//----------------------------------------------------------------------------
// Benchmark Functions
//----------------------------------------------------------------------------


void gpu_convolutionKernelBenchmarkGrayImage(const float *inputImage,
    const float *kernel, float *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, int numKernelTestCalls) {
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  clock_t startTime, endTime;
  float *d_inputImage, *d_kernel, *d_outputImage;
  float fps;

  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;

  assert(kWidth * kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW, BH);
  dim3 gridSize(((iWidth % BW) ? (iWidth / BW + 1) : (iWidth / BW)),
      ((iHeight % BH) ? (iHeight / BH + 1) : (iHeight / BH)));
  int smSize = (blockSize.x + (kRadiusX << 1)) * (blockSize.y + (kRadiusY
      << 1)) * sizeof(float);

  //  allocate memory and copy data
  cutilSafeCall(
      hipMallocPitch((void**) &(d_inputImage), &iPitchBytes,
          iWidth * sizeof(float), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_outputImage), &iPitchBytes,
          iWidth * sizeof(float), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_kernel), &kPitchBytes,
          kWidth * sizeof(float), kHeight));
  iPitch = iPitchBytes / sizeof(float);
  kPitch = kPitchBytes / sizeof(float);

  cutilSafeCall(
      hipMemcpy2D(d_inputImage, iPitchBytes, inputImage,
          iWidth * sizeof(float), iWidth * sizeof(float), iHeight,
          hipMemcpyHostToDevice));
  cutilSafeCall(
      hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth * sizeof(float),
          kWidth * sizeof(float), kHeight, hipMemcpyHostToDevice));

  gpu_bindConstantMemory(kernel, kWidth * kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  // --- global memory only ---
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps << " fps - global memory only\n";

  // --- global memory for image and constant memory for kernel access ---
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps
      << " fps - global memory for image & constant memory for kernel access\n";

  // --- shared memory for image and global memory for kernel access ---
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps
      << " fps - shared memory for image & global memory for kernel access\n";

  // --- shared memory for image and constant memory for kernel access ---
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps
      << " fps - shared memory for image & constant memory for kernel access\n";

  // --- shared memory for image and constant memory for kernel access ---  
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps
      << " fps - dyn. shared memory for image & const memory for kernel access\n";

  // --- texture memory for image and constant memory for kernel access ---
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize>>>(d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC;
  std::cout << fps
      << " fps - texture memory for image & const memory for kernel access\n";

  cutilSafeCall(
      hipMemcpy2D(outputImage, iWidth * sizeof(float), d_outputImage,
          iPitchBytes, iWidth * sizeof(float), iHeight,
          hipMemcpyDeviceToHost));

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall(hipFree(d_inputImage));
  cutilSafeCall(hipFree(d_outputImage));
  cutilSafeCall(hipFree(d_kernel));
}

void gpu_convolutionKernelBenchmarkInterleavedRGB(const float *inputImage,
    const float *kernel, float *outputImage, int iWidth, int iHeight,
    int kRadiusX, int kRadiusY, int numKernelTestCalls) {
  size_t iPitchBytesF3, iPitchBytesF4;
  clock_t startTime, endTime;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  float fps;

  const int kWidth = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;

  assert(kWidth * kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW, BH);
  dim3 gridSize(((iWidth % BW) ? (iWidth / BW + 1) : (iWidth / BW)),
      ((iHeight % BH) ? (iHeight / BH + 1) : (iHeight / BH)));
  int smSizeF3 = (blockSize.x + (kRadiusX << 1)) * (blockSize.y + (kRadiusY
      << 1)) * sizeof(float3);

  //  allocate memory and copy data
  cutilSafeCall(
      hipMallocPitch((void**) &(d_inputImageF3), &iPitchBytesF3,
          iWidth * sizeof(float3), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_inputImageF4), &iPitchBytesF4,
          iWidth * sizeof(float4), iHeight));
  cutilSafeCall(
      hipMallocPitch((void**) &(d_outputImageF3), &iPitchBytesF3,
          iWidth * sizeof(float3), iHeight));
  cutilSafeCall(
      hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage,
          iWidth * sizeof(float3), iWidth * sizeof(float3), iHeight,
          hipMemcpyHostToDevice));

  gpu_bindConstantMemory(kernel, kWidth * kHeight);

  // --- shared memory for interleaved image and constant memory for kernel access ---  
  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC * 3;
  std::cout << fps
      << " fps - dyn. shared mem for interleaved img & const mem for kernel\n";

  // --- texture memory for interleaved image and constant memory for kernel access ---
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);

  startTime = clock();
  for (int c = 0; c < numKernelTestCalls; c++) {
    gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize());
  }
  endTime = clock();
  fps = (float) numKernelTestCalls / float(endTime - startTime)
      * CLOCKS_PER_SEC * 3;
  std::cout << fps
      << " fps - texture mem for interleaved img & const mem for kernel access\n";

  cutilSafeCall(
      hipMemcpy2D(outputImage, iWidth * sizeof(float3), d_outputImageF3,
          iPitchBytesF3, iWidth * sizeof(float3), iHeight,
          hipMemcpyDeviceToHost));

  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall(hipFree(d_inputImageF3));
  cutilSafeCall(hipFree(d_outputImageF3));
  cutilSafeCall(hipFree(d_inputImageF4));
}

void gpu_bindConstantMemory(const float *kernel, int size) {
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(constKernel), kernel, size * sizeof(float)));
}

void gpu_bindTextureMemory(float *d_inputImage, int iWidth, int iHeight,
    size_t iPitchBytes) {
  // >>>> prepare usage of texture memory
  tex_Image.addressMode[0] = hipAddressModeClamp;
  tex_Image.addressMode[1] = hipAddressModeClamp;
  tex_Image.filterMode = hipFilterModeLinear;
  tex_Image.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall(
      hipBindTexture2D(0, &tex_Image, d_inputImage, &tex_Image_desc,
          iWidth, iHeight, iPitchBytes));
}

void gpu_unbindTextureMemory() {
  cutilSafeCall(hipUnbindTexture(tex_Image));
}

void gpu_bindTextureMemoryF4(float4 *d_inputImageF4, int iWidth, int iHeight,
    size_t iPitchBytesF4) {
  // >>>> prepare usage of texture memory
  tex_ImageF4.addressMode[0] = hipAddressModeClamp;
  tex_ImageF4.addressMode[1] = hipAddressModeClamp;
  tex_ImageF4.filterMode = hipFilterModeLinear;
  tex_ImageF4.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall(
      hipBindTexture2D(0, &tex_ImageF4, d_inputImageF4,
          &tex_Image_descF4, iWidth, iHeight, iPitchBytesF4));
}

void gpu_unbindTextureMemoryF4() {
  cutilSafeCall(hipUnbindTexture(tex_ImageF4));
}
