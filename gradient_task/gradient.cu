#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: gradient
* file:    gradient.cu
*
* 
\******* PLEASE ENTER YOUR CORRECT STUDENT LOGIN, NAME AND ID BELOW *********/
const char* studentLogin = "p010";
const char* studentName  = "John Doe";
const int   studentID    = 1234567;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* derivativeY_sm_d(const float *inputImage, ... )
* derivativeY_sm_d(const float3 *inputImage, ... )
* gradient_magnitude_d(const float *inputImage, ... )
* gradient_magnitude_d(const float3 *inputImage, ... )
*
\****************************************************************************/


#include "gradient.cuh"



#define BW 16
#define BH 16



const char* getStudentLogin() { return studentLogin; };
const char* getStudentName()  { return studentName; };
int         getStudentID()    { return studentID; };
bool checkStudentData() { return strcmp(studentLogin, "p010") != 0 && strcmp(studentName, "John Doe") != 0 && studentID != 1234567; };
bool checkStudentNameAndID() { return strcmp(studentName, "John Doe") != 0 && studentID != 1234567; };




__global__ void derivativeX_sm_d(const float *inputImage, float *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float u[BW+2][BH];

  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x);

    if (x == 0) 
      u[threadIdx.x][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (threadIdx.x == 0) 
      u[threadIdx.x][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x-1);

    if (x == (iWidth-1))
      u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (threadIdx.x == blockDim.x-1) 
      u[threadIdx.x+2][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x+1);
  }

  __syncthreads();

  if (x < iWidth && y < iHeight)
    *((float*)(((char*)outputImage) + y*iPitchBytes)+ x) = 0.5f*(u[threadIdx.x+2][threadIdx.y]-u[threadIdx.x][threadIdx.y]);
}




__global__ void derivativeX_sm_d(const float3 *inputImage, float3 *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  float3 imgValue;
  __shared__ float3 u[BW+2][BH];

  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);

    if (x == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (threadIdx.x == 0) u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x-1);
    
    if (x == (iWidth-1)) u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (threadIdx.x == blockDim.x-1) u[threadIdx.x+2][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x+1);
  }

  __syncthreads();

  
  if (x < iWidth && y < iHeight) {
    imgValue.x = 0.5f*(u[threadIdx.x+2][threadIdx.y].x - u[threadIdx.x][threadIdx.y].x);
    imgValue.y = 0.5f*(u[threadIdx.x+2][threadIdx.y].y - u[threadIdx.x][threadIdx.y].y);
    imgValue.z = 0.5f*(u[threadIdx.x+2][threadIdx.y].z - u[threadIdx.x][threadIdx.y].z);
    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = imgValue;
  }
}



__global__ void derivativeY_sm_d(const float *inputImage, float *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float u[BW][BH+2];

  if (x < iWidth && y < iHeight) {
    u[threadIdx.x][threadIdx.y+1] = *((float*)((char*)inputImage + y*iPitchBytes)+x);

    if (y == 0) 
      u[threadIdx.x][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
    else if (threadIdx.y == 0) 
      u[threadIdx.x][threadIdx.y] = *((float*)((char*)inputImage + (y-1)*iPitchBytes)+x);

    if (y == (iHeight-1))
      u[threadIdx.x][threadIdx.y+2] = u[threadIdx.x][threadIdx.y+1];
    else if (threadIdx.y == blockDim.y-1) 
      u[threadIdx.x][threadIdx.y+2] = *((float*)((char*)inputImage + (y+1)*iPitchBytes)+x);
  }

  __syncthreads();

  if (x < iWidth && y < iHeight)
    *((float*)(((char*)outputImage) + y*iPitchBytes)+ x) = 0.5f*(u[threadIdx.x][threadIdx.y+2]-u[threadIdx.x][threadIdx.y]);
}



__global__ void derivativeY_sm_d(const float3 *inputImage, float3 *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
 
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  float3 imgValue;
  __shared__ float3 u[BW][BH+2];

  if (x < iWidth && y < iHeight) {
    u[threadIdx.x][threadIdx.y+1] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);

    if (y == 0) 
      u[threadIdx.x][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
    else if (threadIdx.y == 0) 
      u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + (y-1)*iPitchBytes)+x);
    
    if (y == (iHeight-1)) 
      u[threadIdx.x][threadIdx.y+2] = u[threadIdx.x][threadIdx.y+1];
    else if (threadIdx.y == blockDim.y-1) 
      u[threadIdx.x][threadIdx.y+2] = *((float3*)((char*)inputImage + (y+1)*iPitchBytes)+x);
  }

  __syncthreads();

  
  if (x < iWidth && y < iHeight) {
    imgValue.x = 0.5f*(u[threadIdx.x][threadIdx.y+2].x - u[threadIdx.x][threadIdx.y].x);
    imgValue.y = 0.5f*(u[threadIdx.x][threadIdx.y+2].y - u[threadIdx.x][threadIdx.y].y);
    imgValue.z = 0.5f*(u[threadIdx.x][threadIdx.y+2].z - u[threadIdx.x][threadIdx.y].z);
    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = imgValue;
  }

}


__global__ void gradient_magnitude_d(const float *inputImage, float *outputImage,
                                     int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
 
  __shared__ float u[BW+2][BH+2];
 
  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y+1] = *((float*)((char*)inputImage + y*iPitchBytes)+x);
 
    if (x == 0) 
      u[threadIdx.x][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.x == 0) 
      u[threadIdx.x][threadIdx.y+1] = *((float*)((char*)inputImage + (y)*iPitchBytes)+x-1);
 
    if (x == (iWidth-1))
      u[threadIdx.x+2][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.x == blockDim.x-1) 
      u[threadIdx.x+2][threadIdx.y+1] = *((float*)((char*)inputImage + y*iPitchBytes)+x+1);

    if (y == 0) 
      u[threadIdx.x+1][threadIdx.y] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.y == 0) 
      u[threadIdx.x+1][threadIdx.y] = *((float*)((char*)inputImage + (y-1)*iPitchBytes)+x);

    if (y == (iHeight-1))
      u[threadIdx.x+1][threadIdx.y+2] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.y == blockDim.y-1) 
      u[threadIdx.x+1][threadIdx.y+2] = *((float*)((char*)inputImage + (y+1)*iPitchBytes)+x);
  }
 
  __syncthreads();
 
  //compute the magnitude
  float dIdx, dIdy, magn;

  if (x < iWidth && y < iHeight) {
    dIdx = 0.5f*(u[threadIdx.x+2][threadIdx.y+1]-u[threadIdx.x][threadIdx.y+1]);
    dIdy = 0.5f*(u[threadIdx.x+1][threadIdx.y+2]-u[threadIdx.x+1][threadIdx.y]);
    magn = sqrt(dIdx*dIdx + dIdy*dIdy);
    *((float*)(((char*)outputImage) + y*iPitchBytes)+ x) = magn;
  }
}





__global__ void gradient_magnitude_d(const float3 *inputImage, float3 *outputImage,
                                     int iWidth, int iHeight, size_t iPitchBytes)
{

  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
 
  __shared__ float3 u[BW+2][BH+2];
 
  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y+1] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);
 
    if (x == 0) 
      u[threadIdx.x][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.x == 0) 
      u[threadIdx.x][threadIdx.y+1] = *((float3*)((char*)inputImage + (y)*iPitchBytes)+x-1);
 
    if (x == (iWidth-1))
      u[threadIdx.x+2][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.x == blockDim.x-1) 
      u[threadIdx.x+2][threadIdx.y+1] = *((float3*)((char*)inputImage + y*iPitchBytes)+x+1);

    if (y == 0) 
      u[threadIdx.x+1][threadIdx.y] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.y == 0) 
      u[threadIdx.x+1][threadIdx.y] = *((float3*)((char*)inputImage + (y-1)*iPitchBytes)+x);

    if (y == (iHeight-1))
      u[threadIdx.x+1][threadIdx.y+2] = u[threadIdx.x+1][threadIdx.y+1];
    else if (threadIdx.y == blockDim.y-1) 
      u[threadIdx.x+1][threadIdx.y+2] = *((float3*)((char*)inputImage + (y+1)*iPitchBytes)+x);
  }
 
  __syncthreads();
 
  //compute the magnitude
  float3 dIdx, dIdy, magn;

  if (x < iWidth && y < iHeight) {
    dIdx.x = 0.5f*(u[threadIdx.x+2][threadIdx.y+1].x - u[threadIdx.x][threadIdx.y+1].x);
    dIdx.y = 0.5f*(u[threadIdx.x+2][threadIdx.y+1].y - u[threadIdx.x][threadIdx.y+1].y);
    dIdx.z = 0.5f*(u[threadIdx.x+2][threadIdx.y+1].z - u[threadIdx.x][threadIdx.y+1].z);

    dIdy.x = 0.5f*(u[threadIdx.x+1][threadIdx.y+2].x - u[threadIdx.x+1][threadIdx.y].x);
    dIdy.y = 0.5f*(u[threadIdx.x+1][threadIdx.y+2].y - u[threadIdx.x+1][threadIdx.y].y);
    dIdy.z = 0.5f*(u[threadIdx.x+1][threadIdx.y+2].z - u[threadIdx.x+1][threadIdx.y].z);

    magn.x = sqrt(dIdx.x*dIdx.x + dIdy.x*dIdy.x);
    magn.y = sqrt(dIdx.y*dIdx.y + dIdy.y*dIdy.y);
    magn.z = sqrt(dIdx.z*dIdx.z + dIdy.z*dIdy.z);
    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = magn;
  }

}



void gpu_derivative_sm_d(const float *inputImage, float *outputImage,
                         int iWidth, int iHeight, int iSpectrum, int mode)
{
  size_t iPitchBytes;
  float *inputImage_d = 0, *outputImage_d = 0;

  dim3 blockSize(BW, BH);  
  dim3 gridSize( (int)ceil(iWidth/(float)BW), (int)ceil(iHeight/(float)BH) );
  //dim3 smSize(BW+2,BH);

  if(iSpectrum == 1) {
    cutilSafeCall( hipMallocPitch( (void**)&(inputImage_d), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
    cutilSafeCall( hipMallocPitch( (void**)&(outputImage_d), &iPitchBytes, iWidth*sizeof(float), iHeight ) );

    cutilSafeCall( hipMemcpy2D(inputImage_d, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );

    if (mode == 0)
      derivativeX_sm_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 1)
      derivativeY_sm_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 2)
      gradient_magnitude_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);

    cutilSafeCall( hipDeviceSynchronize() );
    cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), outputImage_d, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );
  }
  else if(iSpectrum == 3) {
    cutilSafeCall( hipMallocPitch( (void**)&(inputImage_d), &iPitchBytes, iWidth*sizeof(float3), iHeight ) );
    cutilSafeCall( hipMallocPitch( (void**)&(outputImage_d), &iPitchBytes, iWidth*sizeof(float3), iHeight ) );

    cutilSafeCall( hipMemcpy2D(inputImage_d, iPitchBytes, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );

    if (mode == 0)
      derivativeX_sm_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 1)
      derivativeY_sm_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 2)
      gradient_magnitude_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);

    cutilSafeCall( hipDeviceSynchronize() );
    cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), outputImage_d, iPitchBytes, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );
  }

  cutilSafeCall( hipFree(inputImage_d) );
  cutilSafeCall( hipFree(outputImage_d) );
}
