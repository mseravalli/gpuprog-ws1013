#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    linearoperations.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */


#include <auxiliary/cuda_basic.cuh>

hipChannelFormatDesc linearoperation_float_tex = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;


#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];


void setTexturesLinearOperations(int mode){
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if(mode == 0)tex_linearoperation.filterMode = hipFilterModePoint;
	else tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}


#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif


#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*)address;
	unsigned int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	}	while (assumed != old);
	return __int_as_float(old);
}

#endif




void backwardRegistrationBilinearValueTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		float value,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me ###
}

void backwardRegistrationBilinearFunctionGlobal
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me ###
}

void backwardRegistrationBilinearFunctionTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me, if you want ###
}




void forewardRegistrationBilinearAtomic
(
		const float *flow1_g,
		const float *flow2_g,
		const float *in_g,
	  float       *out_g,
		int         nx,
		int         ny,
		int         pitchf1
)
{
	// ### Implement me ###
}




void gaussBlurSeparateMirrorGpu
(
		float *in_g,
		float *out_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float sigmax,
		float sigmay,
		int   radius,
		float *temp_g,
		float *mask
)
{
	// ### Implement me ###
}





void resampleAreaParallelSeparate
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}

void resampleAreaParallelSeparateAdjoined
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}


__global__ void addKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	// ### Implement me ###
}

__global__ void subKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	// ### Implement me ###
}

__global__ void setKernel
(
		float *field_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float value
)
{
	// ### Implement me ###
}

