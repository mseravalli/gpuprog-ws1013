#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    flowlib_gpu_sor.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * flowlib_gpu_sor.cu
 *
 *  Created on: Mar 14, 2012
 *      Author: steinbrf
 */

//#include <flowlib_gpu_sor.hpp>
#include "flowlib.hpp"
#include <auxiliary/cuda_basic.cuh>
#include <linearoperations/linearoperations.cuh>
#include <auxiliary/debug.hpp>

hipChannelFormatDesc flow_sor_float_tex = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_flow_sor_I1;
texture<float, 2, hipReadModeElementType> tex_flow_sor_I2;
bool textures_flow_sor_initialized = false;

#define IMAGE_FILTER_METHOD hipFilterModeLinear
#define SF_TEXTURE_OFFSET 0.5f

#define SF_BW 16
#define SF_BH 16


FlowLibGpuSOR::FlowLibGpuSOR(int par_nx, int par_ny):
FlowLib(par_nx,par_ny),FlowLibGpu(par_nx,par_ny),FlowLibSOR(par_nx,par_ny)
{

	cuda_malloc2D((void**)&_penDat,_nx,_ny,1,sizeof(float),&_pitchf1);
	cuda_malloc2D((void**)&_penReg,_nx,_ny,1,sizeof(float),&_pitchf1);

	cuda_malloc2D((void**)&_b1,_nx,_ny,1,sizeof(float),&_pitchf1);
	cuda_malloc2D((void**)&_b2,_nx,_ny,1,sizeof(float),&_pitchf1);

}

FlowLibGpuSOR::~FlowLibGpuSOR()
{
	if(_penDat) cutilSafeCall(hipFree(_penDat));
	if(_penReg) cutilSafeCall(hipFree(_penReg));
	if(_b1)     cutilSafeCall(hipFree(_b1));
	if(_b2)     cutilSafeCall(hipFree(_b2));
}

void bind_textures(const float *I1_g, const float *I2_g, int nx, int ny, int pitchf1)
{
	tex_flow_sor_I1.addressMode[0] = hipAddressModeClamp;
	tex_flow_sor_I1.addressMode[1] = hipAddressModeClamp;
	tex_flow_sor_I1.filterMode = IMAGE_FILTER_METHOD ;
	tex_flow_sor_I1.normalized = false;

	tex_flow_sor_I2.addressMode[0] = hipAddressModeClamp;
	tex_flow_sor_I2.addressMode[1] = hipAddressModeClamp;
	tex_flow_sor_I2.filterMode = IMAGE_FILTER_METHOD;
	tex_flow_sor_I2.normalized = false;

	cutilSafeCall( hipBindTexture2D(0, &tex_flow_sor_I1, I1_g,
		&flow_sor_float_tex, nx, ny, pitchf1*sizeof(float)) );
	cutilSafeCall( hipBindTexture2D(0, &tex_flow_sor_I2, I2_g,
		&flow_sor_float_tex, nx, ny, pitchf1*sizeof(float)) );
}

void unbind_textures_flow_sor()
{
  cutilSafeCall (hipUnbindTexture(tex_flow_sor_I1));
  cutilSafeCall (hipUnbindTexture(tex_flow_sor_I2));
}

void update_textures_flow_sor(const float *I2_resampled_warped_g, int nx_fine, int ny_fine, int pitchf1)
{
	cutilSafeCall (hipUnbindTexture(tex_flow_sor_I2));
	cutilSafeCall( hipBindTexture2D(0, &tex_flow_sor_I2, I2_resampled_warped_g,
		&flow_sor_float_tex, nx_fine, ny_fine, pitchf1*sizeof(float)) );
}


/**
 * @brief Adds one flow field onto another
 * @param du_g Horizontal increment
 * @param dv_g Vertical increment
 * @param u_g Horizontal accumulation
 * @param v_g Vertical accumulation
 * @param nx Image width
 * @param ny Image height
 * @param pitchf1 Image pitch for single float images
 */
__global__ void add_flow_fields
(
	const float *du_g,
	const float *dv_g,
	float *u_g,
	float *v_g,
	int    nx,
	int    ny,
	int    pitchf1
)
{
	// ### Implement Me###
}


/**
 * @brief Kernel to compute the penalty values for several
 * lagged-diffusivity iterations taking into account pixel sizes for warping.
 * Image derivatives are read from texture, flow derivatives from shared memory
 * @param u_g Pointer to global device memory for the horizontal
 * flow component of the accumulation flow field
 * @param v_g Pointer to global device memory for the vertical
 * flow component of the accumulation flow field
 * @param du_g Pointer to global device memory for the horizontal
 * flow component of the increment flow field
 * @param dv_g Pointer to global device memory for the vertical
 * flow component of the increment flow field
 * @param penaltyd_g Pointer to global device memory for data term penalty
 * @param penaltyr_g Pointer to global device memory for regularity term
 * penalty
 * @param nx Image width
 * @param ny Image height
 * @param hx Horizontal pixel size
 * @param hy Vertical pixel size
 * @param data_epsilon Smoothing parameter for the TV Penalization of the data
 * term
 * @param diff_epsilon Smoothing parameter for the TV Penalization of the
 * regularity term
 * @param pitchf1 Image pitch for single float images
 */
__global__ void sorflow_update_robustifications_warp_tex_shared
(
	const float *u_g,
	const float *v_g,
	const float *du_g,
	const float *dv_g,
	float *penaltyd_g,
	float *penaltyr_g,
	int    nx,
	int    ny,
	float  hx,
	float  hy,
	float  data_epsilon,
	float  diff_epsilon,
	int    pitchf1
)
{
	// ### Implement Me###
}


/**
 * @brief Precomputes one value as the sum of all values not depending of the
 * current flow increment
 * @param u_g Pointer to global device memory for the horizontal
 * flow component of the accumulation flow field
 * @param v_g Pointer to global device memory for the vertical
 * flow component of the accumulation flow field
 * @param penaltyd_g Pointer to global device memory for data term penalty
 * @param penaltyr_g Pointer to global device memory for regularity term
 * penalty
 * @param bu_g Pointer to global memory for horizontal result value
 * @param bv_g Pointer to global memory for vertical result value
 * @param nx Image width
 * @param ny Image height
 * @param hx Horizontal pixel size
 * @param hy Vertical pixel size
 * @param lambda Smoothness weight
 * @param pitchf1 Image pitch for single float images
 */
__global__ void sorflow_update_righthandside_shared
(
	const float *u_g,
	const float *v_g,
	const float *penaltyd_g,
	const float *penaltyr_g,
	float *bu_g,
	float *bv_g,
	int    nx,
	int    ny,
	float  hx,
	float  hy,
	float  lambda,
	int    pitchf1
)
{
	// ### Implement Me###
}


/**
 * @brief Kernel to compute one Red-Black-SOR iteration for the nonlinear
 * Euler-Lagrange equation taking into account penalty values and pixel
 * size for warping
 * @param bu_g Right-Hand-Side values for horizontal flow
 * @param bv_g Right-Hand-Side values for vertical flow
 * @param penaltyd_g Pointer to global device memory holding data term penalization
 * @param penaltyr_g Pointer to global device memory holding regularity term
 * penalization
 * @param du_g Pointer to global device memory for the horizontal
 * flow component increment
 * @param dv_g Pointer to global device memory for the vertical
 * flow component increment
 * @param nx Image width
 * @param ny Image height
 * @param hx Horizontal pixel size
 * @param hy Vertical pixel size
 * @param lambda Smoothness weight
 * @param relaxation Overrelaxation for the SOR-solver
 * @param red Parameter deciding whether the red or black fields of a
 * checkerboard pattern are being updated
 * @param pitchf1 Image pitch for single float images
 */
__global__ void sorflow_nonlinear_warp_sor_shared
(
	const float *bu_g,
	const float *bv_g,
	const float *penaltyd_g,
	const float *penaltyr_g,
	float *du_g,
	float *dv_g,
	int    nx,
	int    ny,
	float  hx,
	float  hy,
	float  lambda,
	float  relaxation,
	int    red,
	int    pitchf1
)
{
	// ### Implement Me ###
}

/**
 * @brief Method that calls the sorflow_nonlinear_warp_sor_shared in a loop,
 * with an outer loop for computing the diffisivity values for
 * one level of a coarse-to-fine implementation.
 * @param u_g Pointer to global device memory for the horizontal
 * flow component
 * @param v_g Pointer to global device memory for the vertical
 * flow component
 * @param du_g Pointer to global device memory for the horizontal
 * flow component increment
 * @param dv_g Pointer to global device memory for the vertical
 * flow component increment
 * @param bu_g Right-Hand-Side values for horizontal flow
 * @param bv_g Right-Hand-Side values for vertical flow
 * @param penaltyd_g Pointer to global device memory holding data term penalization
 * @param penaltyr_g Pointer to global device memory holding regularity term
 * penalization
 * @param nx Image width
 * @param ny Image height
 * @param pitchf1 Image pitch for single float images
 * @param hx Horizontal pixel size
 * @param hy Vertical pixel size
 * @param lambda Smoothness weight
 * @param outer_iterations Number of iterations of the penalty computation
 * @param inner_iterations Number of iterations for the SOR-solver
 * @param relaxation Overrelaxation for the SOR-solver
 * @param data_epsilon Smoothing parameter for the TV Penalization of the data
 * term
 * @param diff_epsilon Smoothing parameter for the TV Penalization of the
 * regularity term
 */
void sorflow_gpu_nonlinear_warp_level
(
		const float *u_g,
		const float *v_g,
		float *du_g,
		float *dv_g,
		float *bu_g,
		float *bv_g,
		float *penaltyd_g,
		float *penaltyr_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float hx,
		float hy,
		float lambda,
		float overrelaxation,
		int   outer_iterations,
		int   inner_iterations,
		float data_epsilon,
		float diff_epsilon
)
{
	// ### Implement Me ###
}


float FlowLibGpuSOR::computeFlow()
{
	// ### Implement Me###
}

